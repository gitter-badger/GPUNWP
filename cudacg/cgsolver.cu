#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include<string.h>





#include<hip/hip_runtime_api.h>
#include<hipsparse.h>
#include<hipblas.h>
#include"CG.h"



int gpu_solver(const struct N n, const REAL *b, const REAL* x, REAL resreduction){
  unsigned int maxiter=20;
  unsigned int k;
  unsigned long n_lin=n.x*n.y*n.z;
  REAL* r;
  REAL* z;
  REAL* p;
  REAL* q; 
  REAL* tq;
  const REAL negone=-1.0;
  N *dev_n,*l_n;
  hipHostMalloc(&r, n_lin*sizeof(REAL));
  hipHostMalloc(&z, n_lin*sizeof(REAL));
  hipHostMalloc(&p, n_lin*sizeof(REAL));
  hipHostMalloc(&q, n_lin*sizeof(REAL));
  hipHostMalloc(&l_n, n_lin*sizeof(N));
  hipHostMalloc(&tq,n_lin*sizeof(REAL));//for test qpply function
  REAL alpha, beta, temp;
  REAL rnorm, rnorm0, rnorm_old,rz, rznew;
 
  

  //GPU Memory Allocation
  REAL *dev_x,*dev_b,*dev_r,*dev_z,*dev_p,*dev_q;
  hipMalloc((void**)&dev_x,n_lin*sizeof(REAL));  
  hipMalloc((void**)&dev_b,n_lin*sizeof(REAL));
  hipMalloc((void**)&dev_r,n_lin*sizeof(REAL));  
  hipMalloc((void**)&dev_z,n_lin*sizeof(REAL)); 
  hipMalloc((void**)&dev_p,n_lin*sizeof(REAL));  
  hipMalloc((void**)&dev_q,n_lin*sizeof(REAL)); 
  hipMalloc((void**)&dev_n,sizeof(N));



  //Memory copy

  l_n->x=n.x;
  l_n->y=n.y;
  l_n->z=n.z;

  

  hipMemcpy(dev_n,l_n,sizeof(N),hipMemcpyHostToDevice);
  hipMemcpy(dev_x,x,n_lin*sizeof(REAL),hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,n_lin*sizeof(REAL),hipMemcpyHostToDevice);
 
  //Initialise CUBLAS
  hipblasHandle_t cublasHandle=0;
  hipblasCreate(&cublasHandle);

  /*Initilise CG solver (Iteration 0)*/
  hipblasScopy(cublasHandle,n_lin,dev_b,1,dev_r,1);
  gpu_apply(l_n,dev_x,dev_q);
  

  hipblasSaxpy(cublasHandle,n_lin,&negone,dev_q,1,dev_r,1);//r_0=b_0-Ax_0
  if(use_prec){
    // gpu_bj(l_n,dev_r,dev_z);
  }
  else
    hipblasScopy(cublasHandle,n_lin,dev_r,1,dev_z,1);
  hipblasScopy(cublasHandle,n_lin,dev_z,1,dev_p,1);//r_0->p_0
  hipblasSdot(cublasHandle,n_lin,dev_r,1,dev_z,1,&rz);
  hipblasSnrm2(cublasHandle,n_lin,dev_r,1,&rnorm0);
  rnorm_old=rnorm0;

  printf("CG initial residual %8.4e\n",rnorm0);

  /*
   *CG Iteration
   */
  for(k=1;k<2;k++){

    gpu_apply(l_n,dev_p,dev_q);
    ///////////////////////////////////////////////////////////////////////////
    hipMemcpy(p,dev_p,n_lin*sizeof(REAL),hipMemcpyDeviceToHost);
    hipMemcpy(q,dev_q,n_lin*sizeof(REAL),hipMemcpyDeviceToHost);
    apply(n,p,tq);
    int err=0;
    int j;
    for(j=0;j<n_lin;j++){
      if(tq[j]!=q[j]){
	err++;
     	printf("%f---%f\n",tq[j],q[j]);
      }
    }
      printf("apply(%d) error=%d\n",k,err);
      /////////////////////////////////////////////////////////////////////////
    hipblasSdot(cublasHandle,n_lin,dev_p,1,dev_q,1,&temp);
    alpha=rz/temp;
    float negalpha=0-alpha;
    hipblasSaxpy(cublasHandle,n_lin,&alpha,dev_p,1,dev_x,1);
    hipblasSaxpy(cublasHandle,n_lin,&negalpha,dev_q,1,dev_r,1);
    hipblasSnrm2(cublasHandle,n_lin,dev_r,1,&rnorm);

    //    printf("iteration %d||r||=%8.3e rho_r=%6.3f, beta=%f, alpha=%f\n",k,rnorm,rnorm/rnorm_old,beta,alpha);

    if(rnorm/rnorm0<resreduction) break;

    if(use_prec){
      //gpu_bj(l_n,dev_r,dev_z);
    }
    else
      hipblasScopy(cublasHandle,n_lin,dev_r,1,dev_z,1);
    hipblasSdot(cublasHandle,n_lin,dev_r,1,dev_z,1,&rznew);
    beta=rznew/rz;
    hipblasSaxpy(cublasHandle,n_lin,&beta,dev_z,1,dev_p,1);
    rz=rznew;
    rnorm_old=rnorm;
  }


  hipblasDestroy(cublasHandle);
  hipFree(dev_r);
  hipFree(dev_z);
  hipFree(dev_p);
  hipFree(dev_q);

  return 0;

}

